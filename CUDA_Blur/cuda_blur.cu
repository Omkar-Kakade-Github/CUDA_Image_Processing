#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// CUDA kernel for blurring an RGB image
__global__ void cuda_blur(unsigned char *input_image, unsigned char *output_image, int image_width, int image_height, int blur_radius) {
    // Calculate the pixel coordinates this thread is responsible for
    int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the pixel coordinates are within the image bounds
    if (pixel_x >= image_width || pixel_y >= image_height) {
        return;
    }

    // Variables to accumulate the sum of RGB values in the neighborhood
    int red_sum = 0;
    int green_sum = 0;
    int blue_sum = 0;
    int pixel_count = 0;

    // Iterate over the neighborhood defined by the blur radius
    for (int offset_y = -blur_radius; offset_y <= blur_radius; offset_y++) {
        for (int offset_x = -blur_radius; offset_x <= blur_radius; offset_x++) {
            // Calculate the coordinates of the neighboring pixel
            int neighbor_x = pixel_x + offset_x;
            int neighbor_y = pixel_y + offset_y;

            // Check if the neighboring pixel is within the image bounds
            if (neighbor_x >= 0 && neighbor_x < image_width && neighbor_y >= 0 && neighbor_y < image_height) {
                // Calculate the index of the neighboring pixel in the input image array
                int neighbor_index = (neighbor_y * image_width + neighbor_x) * 3;

                // Accumulate the RGB values of the neighboring pixel
                red_sum += input_image[neighbor_index];
                green_sum += input_image[neighbor_index + 1];
                blue_sum += input_image[neighbor_index + 2];
                pixel_count++;
            }
        }
    }

    // Calculate the index of the current pixel in the output image array
    int output_index = (pixel_y * image_width + pixel_x) * 3;

    // Compute the average RGB values and write them to the output image
    output_image[output_index] = red_sum / pixel_count;         // Red channel
    output_image[output_index + 1] = green_sum / pixel_count;   // Green channel
    output_image[output_index + 2] = blue_sum / pixel_count;    // Blue channel
}

int main() {
    // Load the input image
    int width, height, channels;
    unsigned char *input_image = stbi_load("/home/omkar/Brendan/Projects/CUDA_Blur/input.jpg", &width, &height, &channels, 3);
    if (!input_image) {
        std::cerr << "Error: Could not load image!" << std::endl;
        return -1;
    }

    // Allocate memory for the output image
    unsigned char *output_image = (unsigned char *)malloc(width * height * 3);
    if (!output_image) {
        std::cerr << "Error: Could not allocate memory for output image!" << std::endl;
        stbi_image_free(input_image);
        return -1;
    }

    // Allocate device memory
    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, width * height * 3);
    hipMalloc((void **)&d_output, width * height * 3);

    // Copy input image to device
    hipMemcpy(d_input, input_image, width * height * 3, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Set blur radius
    int radius = 5;

    // Launch the CUDA kernel
    cuda_blur<<<gridDim, blockDim>>>(d_input, d_output, width, height, radius);

    // Copy the result back to the host
    hipMemcpy(output_image, d_output, width * height * 3, hipMemcpyDeviceToHost);

    // Save the output image
    stbi_write_jpg("output_blurred.jpg", width, height, 3, output_image, 100);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    stbi_image_free(input_image);
    free(output_image);

    std::cout << "Blurred image saved as output_blurred.jpg" << std::endl;
    return 0;
}
